#include "hip/hip_runtime.h"
#include "iostream"
#include <chrono>
#include "Eigen/Dense"
#include <cstdio>
#include <cstdlib>
#include <cassert>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define CUDAErrorCheck(cudaStatus, customMSG) (CUDAErrorCheck(cudaStatus, __FILE__, __LINE__, customMSG))
#define CUBLASErrorCheck(hipblasStatus_t, customMSG) (CUBLASErrorCheck(hipblasStatus_t, __FILE__, __LINE__, customMSG))

typedef Eigen::MatrixXcd MatriX;
void CUBLASZgemm(const Eigen::Ref<const MatriX> &A, const Eigen::Ref<const MatriX> &B, Eigen::Ref<MatriX> C,
                 const MatriX::Scalar &alpha = 1.0, const MatriX::Scalar &beta = 0.0){
    static_assert(std::is_same<MatriX::Scalar, std::complex<double>>::value);
    static_assert(!MatriX::IsRowMajor, "CUDA::CUBLASZgemm(A, B, C,...) requires column major storage of A, B, and C.");

    // CUDA and cuBLAS status and handle
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    // Host (CPU) pointers of A, B, and C typed in CUDA's custom implementation of std::complex double
    const hipDoubleComplex *pAHost = nullptr;
    const hipDoubleComplex *pBHost = nullptr;
    hipDoubleComplex *pCHost = nullptr;

    // Device (GPU) pointers of A, B, and C
    const hipDoubleComplex *devPtrA = nullptr;
    const hipDoubleComplex *devPtrB = nullptr;
    hipDoubleComplex *devPtrC = nullptr;
    std::complex<double> *devPtrCInSTDComplexDouble = nullptr;

    // Clean up
    auto BurnItAll = [&devPtrA, &devPtrB, &devPtrC, &handle] (){
        hipFree(devPtrA);
        hipFree(devPtrB);
        hipFree(devPtrC);
        hipblasDestroy(handle);
    };

    // Grab dimensions of A, B, and C
    const Eigen::Index rowsOfA = A.rows();
    const Eigen::Index colsOfA = A.cols();
    const Eigen::Index sizeOfA = A.size();
    const Eigen::Index &ldA = rowsOfA; // Column major storage enforced above

    const Eigen::Index rowsOfB = B.rows();
    const Eigen::Index colsOfB = B.cols();
    const Eigen::Index sizeOfB = B.size();
    const Eigen::Index &ldB = rowsOfB; // Column major storage enforced above

    const Eigen::Index rowsOfC = C.rows();
    const Eigen::Index colsOfC = C.cols();
    const Eigen::Index sizeOfC = C.size();
    const Eigen::Index &ldC = rowsOfC; // Column major storage enforced above

    assert(colsOfA == rowsOfB);
    assert(rowsOfA == rowsOfC);
    assert(colsOfB == colsOfC);

    // Retrieve underlying data from Eigen, then cast std::complex double into hipDoubleComplex
    pAHost = reinterpret_cast<const hipDoubleComplex*>(A.data());
    pBHost = reinterpret_cast<const hipDoubleComplex*>(B.data());
    pCHost = reinterpret_cast<hipDoubleComplex*>(C.data());

    // Cast alpha and beta into hipDoubleComplex
    const hipDoubleComplex *pAlpha = reinterpret_cast<hipDoubleComplex*>(&alpha);
    const hipDoubleComplex *pBeta = reinterpret_cast<hipDoubleComplex*>(&beta);

    // Destroy allocated
    BurnItAll();
}

int main(){
    //  Initializing Eigen Matrices
    int N = 500;
    MatriX a_E = MatriX::Identity(N,N);
    MatriX b_E = MatriX::Ones(N,N);
    MatriX c_E = MatriX::Zero(N,N);

    auto startDot = std::chrono::high_resolution_clock::now();
    c_E.noalias() = a_E * b_E;
    auto stopDot = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> durationDot = stopDot - startDot;
    std::cout<<"duration: " <<durationDot.count()<<'\n';
    std::cout<<"||c_E||: " <<c_E.norm()<<'\n';

    MultiplyUsingCUBLAS(a_E, b_E, c_E);

    //std::cout<<"a_E: " <<a_E<<'\n';
    //std::cout<<"b_E: " <<b_E<<'\n';
    //unsigned a = 4u;
    //unsigned b = 4u;
    //Eigen::MatrixXcd hi = Eigen::MatrixXcd::Zero(a,b);
}
